#include "hip/hip_runtime.h"
#include <cstdint>
#include <cstdio>
#include <ctime>
#include <cstring>
#include <glm/gtc/type_ptr.hpp>

#include "volrend/n3tree.hpp"
#include "volrend/cuda/rt_naive.cuh"
#include "volrend/render_options.hpp"

namespace volrend {

#define MAX3(a, b, c) max(max(a, b), c)
#define MIN3(a, b, c) min(min(a, b), c)

// __host__ __device__ __inline__ static void world2screen(
//         const float* __restrict__ xyz,
//         float focal_x,
//         float focal_y,
//         const float* __restrict__ transform,
//         float* out) {
//     float x, y, z;
//     x = xyz[0] - transform[9];
//     y = xyz[1] - transform[10];
//     z = xyz[2] - transform[11];
//     float zt;
//     out[0] = transform[0] * x + transform[1] * y + transform[2] * z;
//     out[1] = transform[3] * x + transform[4] * y + transform[5] * z;
//     zt     = transform[6] * x + transform[7] * y + transform[8] * z;
//
//     out[0] *= focal_x / zt;
//     out[1] *= focal_y / zt;
// }

__host__ __device__ __inline__ static void screen2worlddir(
        float x, float y, float focal_norm_x,
        float focal_norm_y,
        const float* __restrict__ transform,
        float* out) {
    x /= focal_norm_x;
    y /= focal_norm_y;
    float z = sqrtf(x * x + y * y + 1.0);
    x /= z;
    y /= z;
    z = 1.0f / z;

    out[0] = transform[0] * x + transform[3] * y + transform[6] * z;
    out[1] = transform[1] * x + transform[4] * y + transform[7] * z;
    out[2] = transform[2] * x + transform[5] * y + transform[8] * z;
}

namespace device {
// Primary rendering kernel
__global__ static void render_kernel(
        hipSurfaceObject_t surf_obj,
        const int width,
        const int height,
        float focal_norm_x,
        float focal_norm_y,
        const float* __restrict__ transform,
        const float* __restrict__ tree_data,
        const int32_t* __restrict__ tree_child,
        int tree_N,
        float step_size,
        float stop_thresh,
        float background_brightness) {
    CUDA_GET_THREAD_ID(idx, width * height);
    const int x   = idx % width;
    const int y   = idx / width;

    const float x_norm = x / (0.5f * width) - 1.0f;
    const float y_norm = y / (0.5f * height) - 1.0f;

    float dir[3], out[3];
    screen2worlddir(x_norm, y_norm, focal_norm_x, focal_norm_y, transform, dir);

    trace_ray_naive(tree_data, tree_child, tree_N,
            dir, transform + 9, step_size, stop_thresh, background_brightness, out);

    // pixel color
    uint8_t rgbx[4];
    rgbx[0]  = uint8_t(out[0] * 255);
    rgbx[1] = uint8_t(out[1] * 255);
    rgbx[2]  = uint8_t(out[2] * 255);
    rgbx[3] = 255;

    surf2Dwrite(
            *reinterpret_cast<uint32_t*>(rgbx),
            surf_obj,
            x * 4,
            y,
            hipBoundaryModeZero); // squelches out-of-bound writes
}

}  // namespace device

__host__ void launch_renderer(const N3Tree& tree,
        const Camera& cam, const RenderOptions& options, hipArray_t& arr,
        hipStream_t stream) {
    struct hipResourceDesc res_desc;

    // Init surface object
    memset(&res_desc, 0, sizeof(res_desc));
    res_desc.resType = hipResourceTypeArray;
    res_desc.res.array.array = arr;

    hipSurfaceObject_t surf_obj = 0;
    hipCreateSurfaceObject(&surf_obj, &res_desc);

    const int blocks = N_BLOCKS_NEEDED(cam.width * cam.height);
    float focal_norm_x = cam.focal / (cam.width * 0.5f);
    float focal_norm_y = cam.focal / (cam.height * 0.5f);

    device::render_kernel<<<blocks, N_CUDA_THREADS, 0, stream>>>(
            surf_obj,
            cam.width, cam.height,
            focal_norm_x, focal_norm_y, cam.device.transform,
            tree.device.data,
            tree.device.child,
            tree.N,
            options.step_size,
            options.stop_thresh,
            options.background_brightness);
}
}  // namespace volrend
