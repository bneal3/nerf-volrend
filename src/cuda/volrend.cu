#include "hip/hip_runtime.h"
#include <cstdint>
#include <cstdio>
#include <ctime>
#include <cstring>
#include <glm/gtc/type_ptr.hpp>

#include "volrend/n3tree.hpp"
#include "volrend/cuda/rt_core.cuh"
#include "volrend/render_options.hpp"

namespace volrend {

#define MAX3(a, b, c) max(max(a, b), c)
#define MIN3(a, b, c) min(min(a, b), c)

// __host__ __device__ __inline__ static void world2screen(
//         const float* __restrict__ xyz,
//         float focal_x,
//         float focal_y,
//         const float* __restrict__ transform,
//         float* out) {
//     float x, y, z;
//     x = xyz[0] - transform[9];
//     y = xyz[1] - transform[10];
//     z = xyz[2] - transform[11];
//     float zt;
//     out[0] = transform[0] * x + transform[1] * y + transform[2] * z;
//     out[1] = transform[3] * x + transform[4] * y + transform[5] * z;
//     zt     = transform[6] * x + transform[7] * y + transform[8] * z;
//
//     out[0] *= focal_x / zt;
//     out[1] *= focal_y / zt;
// }

template<typename scalar_t>
__host__ __device__ __inline__ static scalar_t _norm(
        scalar_t* dir) {
    return sqrtf(dir[0] * dir[0] + dir[1] * dir[1] + dir[2] * dir[2]);
}

template<typename scalar_t>
__host__ __device__ __inline__ static void _normalize(
        scalar_t* dir) {
    scalar_t norm = _norm(dir);
    dir[0] /= norm; dir[1] /= norm; dir[2] /= norm;
}

__host__ __device__ __inline__ static void screen2worlddir(
        int ix, int iy, float focal,
        int width, int height,
        const float* __restrict__ transform,
        float* out,
        float* cen) {
    float x = (ix - 0.5 * width) / focal;
    float y = -(iy - 0.5 * height) / focal;
    float z = sqrtf(x * x + y * y + 1.0);
    x /= z;
    y /= z;
    z = -1.0f / z;

    out[0] = transform[0] * x + transform[3] * y + transform[6] * z;
    out[1] = transform[1] * x + transform[4] * y + transform[7] * z;
    out[2] = transform[2] * x + transform[5] * y + transform[8] * z;
    cen[0] = transform[9]; cen[1] = transform[10]; cen[2] = transform[11];
}
template<typename scalar_t>
__host__ __device__ __inline__ void world2ndc(
        int ndc_width, int ndc_height, scalar_t ndc_focal,
        scalar_t* __restrict__ dir,
        scalar_t* __restrict__ cen, scalar_t near = 1.f) {
    scalar_t t = -(near + cen[2]) / dir[2];
    for (int i = 0; i < 3; ++i) {
        cen[i] = cen[i] + t * dir[i];
    }

    dir[0] = -((2 * ndc_focal) / ndc_width) * (dir[0] / dir[2] - cen[0] / cen[2]);
    dir[1] = -((2 * ndc_focal) / ndc_height) * (dir[1] / dir[2] - cen[1] / cen[2]);
    dir[2] = -2 * near / cen[2];

    cen[0] = -((2 * ndc_focal) / ndc_width) * (cen[0] / cen[2]);
    cen[1] = -((2 * ndc_focal) / ndc_height) * (cen[1] / cen[2]);
    cen[2] = 1 + 2 * near / cen[2];

    _normalize(dir);
}

template <typename scalar_t>
__device__ __inline__ scalar_t _get_delta_scale(
    const scalar_t* __restrict__ scaling,
    scalar_t* __restrict__ dir) {
    dir[0] *= scaling[0];
    dir[1] *= scaling[1];
    dir[2] *= scaling[2];
    scalar_t delta_scale = 1.f / _norm(dir);
    dir[0] *= delta_scale;
    dir[1] *= delta_scale;
    dir[2] *= delta_scale;
    return delta_scale;
}

namespace device {
// Primary rendering kernel
__global__ static void render_kernel(
        hipSurfaceObject_t surf_obj,
        const int width,
        const int height,
        float focal,
        const float* __restrict__ transform,
        const float* __restrict__ tree_data,
        const int32_t* __restrict__ tree_child,
        const float* __restrict__ tree_offset,
        const float* __restrict__ tree_scale,
        int tree_N,
        int data_dim,
        int sh_order,
        float ndc_width,
        float ndc_height,
        float ndc_focal,
        float step_size,
        float stop_thresh,
        float sigma_thresh,
        float background_brightness,
        bool show_cuda) {
    CUDA_GET_THREAD_ID(idx, width * height);
    const int x   = idx % width;
    const int y   = idx / width;
    // if (x > 0 || y > 0) return;

    float dir[3], cen[3], out[3];
    screen2worlddir(x, y, focal, width, height, transform, dir,
            cen);
    float vdir[3] = {dir[0], dir[1], dir[2]};
    if (ndc_width > 0.f) {
        world2ndc(ndc_width, ndc_height, ndc_focal, dir, cen);
    }
    for (int i = 0; i < 3; ++i) {
        cen[i] = tree_offset[i] + tree_scale[i] * cen[i];
    }

    const float delta_scale = _get_delta_scale(tree_scale, dir);
    trace_ray(tree_data, tree_child, tree_N, data_dim, sh_order,
            dir,
            vdir,
            cen, step_size, stop_thresh,
            sigma_thresh,
            background_brightness,
            delta_scale,
            show_cuda, out);

    // pixel color
    uint8_t rgbx[4];
    rgbx[0]  = uint8_t(out[0] * 255);
    rgbx[1] = uint8_t(out[1] * 255);
    rgbx[2]  = uint8_t(out[2] * 255);
    rgbx[3] = 255;

    surf2Dwrite(
            *reinterpret_cast<uint32_t*>(rgbx),
            surf_obj,
            x * 4,
            y,
            hipBoundaryModeZero); // squelches out-of-bound writes
}

}  // namespace device

__host__ void launch_renderer(const N3Tree& tree,
        const Camera& cam, const RenderOptions& options, hipArray_t& arr,
        hipStream_t stream) {
    struct hipResourceDesc res_desc;

    // Init surface object
    memset(&res_desc, 0, sizeof(res_desc));
    res_desc.resType = hipResourceTypeArray;
    res_desc.res.array.array = arr;

    hipSurfaceObject_t surf_obj = 0;
    hipCreateSurfaceObject(&surf_obj, &res_desc);

    // 128 is weirdly faster for me than 1024
    // Not sure if this scales to a good GPU
    const int N_CUDA_THREADS = 128;

    const int blocks = N_BLOCKS_NEEDED(cam.width * cam.height, N_CUDA_THREADS);
    device::render_kernel<<<blocks, N_CUDA_THREADS, 0, stream>>>(
            surf_obj,
            cam.width, cam.height,
            cam.focal, cam.device.transform,
            tree.device.data,
            tree.device.child,
            tree.device.offset,
            tree.device.scale,
            tree.N,
            tree.data_dim,
            tree.sh_order,
            tree.use_ndc ? tree.ndc_width : -1,
            tree.ndc_height,
            tree.ndc_focal,
            options.step_size,
            options.stop_thresh,
            options.sigma_thresh,
            options.background_brightness,
            options.show_grid);
}
}  // namespace volrend
